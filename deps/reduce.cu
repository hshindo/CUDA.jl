
#include <hip/hip_runtime.h>
__inline__ __device__ int warpReduceSum(int val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        val += __shfl_down(val, offset);
    }
    return val;
}

__global__ void reduce(int *in, int *out, int N) {
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < N;
        i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    //sum = warpReduceSum(sum);
    //if (threadIdx.x & (warpSize - 1) == 0) atomicAdd(out, sum);
}
